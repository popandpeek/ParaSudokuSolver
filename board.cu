#include "hip/hip_runtime.h"
/*
** Ben Pittman
** Greg Smith
** Calvin Winston Fei
** Term Project - board.cpp
** Static class for checking solutions.
** Assumptions: Assumes valid board size and 1D memory allocation
*/

#include <stdio.h>
#include <iostream>
#include <set>
#include "hip/hip_runtime.h"
#include ""

const int BOARD_SIZE = 81;
const int SUB_BOARD_SIZE = 9;

class Board {

public:

	// Array of bool pointers to hold cells for board
	// 0 item in each array signifies filled or empty, 1-9 signifies filled value or potential value
	bool **board[BOARD_SIZE];

	__host__ Board() {
		for (int i = 0; i < BOARD_SIZE; i++) {
			*board[i] = (bool*)malloc((SUB_BOARD_SIZE + 1) * sizeof(bool));
		}
	};

	// Method to set the board according to passed integer array
	// assumes the filled integer array is of size BOARD_SIZE contains only values between 1 and 9
	__host__ void set_board(int* filled) {
		for (int i = 0; i < BOARD_SIZE; i++) {
			if (filled[i] != 0) {
				*board[i][0] = true;
				*board[i][filled[i]] = true;
			}
		}
	}

	// sets a cell as 
	__device__ void set_cell(int _row, int _col, int _val) {
		int board_cell = _row + _col * SUB_BOARD_SIZE;
		*board[board_cell][0] = true;
		for (int i = 1; i < SUB_BOARD_SIZE + 1; i++) {
			if (*board[board_cell][i] == true && i != _val) {
				*board[board_cell][0] = false;
			}
		}
	}

	// method for finding potential values for empty cells
	__device__ void annotate_potential_entries(int row, int col) {
		// scan row for values and store in temp set
		std::set<int> row_vals;
		std::set<int> col_vals;

		// find filled cells in the row and add to array
		for (int i = row * SUB_BOARD_SIZE; i < (row * SUB_BOARD_SIZE) + SUB_BOARD_SIZE; i++) {
			if (*board[i][0]) {
				for (int j = 1; j < SUB_BOARD_SIZE; j++) {
					if (*board[i][j]) {
						row_vals.insert(j);
					}
				}
			}
		}

		// scan col for values and store in temp set
		for (int i = col; i < col * SUB_BOARD_SIZE - col; i += SUB_BOARD_SIZE) {
			if (*board[i][0]) {
				for (int j = 1; j < SUB_BOARD_SIZE; j++) {
					if (*board[i][j]) {
						col_vals.insert(j);
					}
				}
			}
		}

		// Fill potential entries in rows
		if (!row_vals.empty()) {
			for (int i = row * SUB_BOARD_SIZE; i < (row * SUB_BOARD_SIZE) + SUB_BOARD_SIZE; i++) {
				if (!*board[i][0]) {
					for (auto it = row_vals.begin(); it != row_vals.end(); ++it) {
						*board[i][*it] = true;
					}
				}
			}
		}

		// Fill potential entries in columns
		if (!col_vals.empty()) {
			for (int i = col; i < col * BOARD_SIZE - col; i += SUB_BOARD_SIZE) {
				if (*board[i][0]) {
					for (auto it = col_vals.begin(); it != col_vals.end(); ++it) {
						*board[i][*it] = true;
					}
				}
			}
		}
	}

	// Prints out the passed in sudoku game board
	// Assumes N is either 4, 9 or 16 but can be extended to add more sizes
	__host__ void print_board(int *sudoku) {

		char* border;
		if (SUB_BOARD_SIZE == 4) {
			border = new char[14]{ "|-----+-----|" };
		}
		else if (SUB_BOARD_SIZE == 9) {
			border = new char[26]{ "|-------+-------+-------|" };
		}
		else if (SUB_BOARD_SIZE == 16) {
			border = new char[42]{ "|---------+---------+---------+---------|" };
		}
		else {
			return;
		}

		std::cout << border << std::endl;
		int split = sqrt(SUB_BOARD_SIZE);
		for (int i = 0; i < SUB_BOARD_SIZE*SUB_BOARD_SIZE; i++) {
			if (i % SUB_BOARD_SIZE == 0) {
				std::cout << "| ";
			}
			else if (i % split == 0) {
				std::cout << "| ";
			}
			
			// change to call a get_entry fucntion that will return the value
			int value = sudoku[i];
			if (value != 0) {
				std::cout << value << " ";
			}
			else {
				std::cout << ". ";
			}

			if (i % SUB_BOARD_SIZE == SUB_BOARD_SIZE - 1) {
				std::cout << "|" << std::endl;

				if (((i + 1) % (SUB_BOARD_SIZE * SUB_BOARD_SIZE / split)) == 0) {
					std::cout << border << std::endl;
				}
			}
		}
		std::cout << std::endl;
	}
};
